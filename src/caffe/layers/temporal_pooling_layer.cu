#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

using std::min;
using std::max;

template <typename Dtype>
void TemporalPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int top_count = (*top)[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  //const bool use_top_mask = top->size() > 1;
  int* mask = NULL;  // suppress warnings about uninitalized variables
  //Dtype* top_mask = NULL;
  int offset = vl_ * width_ * height_;
  // Different pooling methods. We explicitly do the switch outside the for
  // loop to save time, although this results in more code.
  switch (this->layer_param_.temporal_pooling_param().pool()) {
  case TemporalPoolingParameter_PoolMethod_MAX:
    // Initialize
    mask = max_idx_.mutable_gpu_data();
    caffe_gpu_set(top_count, -1, mask);
    caffe_gpu_set(top_count, Dtype(-FLT_MAX), top_data);
    // The main loop
    for (int n = 0; n < bottom[0]->num(); ++n) {
      for (int g = 0; g < pooled_length_; g++) {
        int offset_ng = (*top)[0]->offset(n) + offset * g;
        if (g * stride_ < pad_) {
          int valid_count = kernel_size_ - pad_ + g * stride_;  // last #valid_count input inside the kernel_size
          for (int index = 0; index < valid_count; index ++) {
            caffe_gpu_vimax(offset, top_data + offset_ng, mask + offset_ng, 
                            bottom_data + bottom[0]->offset(n) + index * offset, 
                            index);
          }
        } else {
          int valid_count = min(kernel_size_, pad_ + group_ - g * stride_);  
          for (int index = 0; index < valid_count; index++) {
            caffe_gpu_vimax(offset, top_data + offset_ng, mask + offset_ng, 
                            bottom_data + bottom[0]->offset(n) + (g * stride_ - pad_ + index) * offset, 
                            index);
          }
        }
      }
    }
    break;
  case TemporalPoolingParameter_PoolMethod_AVE:
    NOT_IMPLEMENTED;
    break;
  case TemporalPoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
}

template <typename Dtype>
void TemporalPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  // Different pooling methods. We explicitly do the switch outside the for
  // loop to save time, although this results in more codes.
  caffe_gpu_set((*bottom)[0]->count(), Dtype(0), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  //const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;  // suppress warnings about uninitialized variables
  int offset = vl_ * width_ * height_;
  //const Dtype* top_mask = NULL;
  switch (this->layer_param_.temporal_pooling_param().pool()) {
  case TemporalPoolingParameter_PoolMethod_MAX:
    // The main loop
    //if (use_top_mask) {
    //  top_mask = top[1]->gpu_data();
    //} else {
    mask = max_idx_.gpu_data();
    //}
    for (int n = 0; n < (*bottom)[0]->num(); ++n) {
      for (int g = 0; g < pooled_length_; g++) {
        int offset_ng = top[0]->offset(n) + offset * g;
        if (g * stride_ < pad_) {
          caffe_gpu_backfill(offset, top_diff + offset_ng, mask + offset_ng, 
                            bottom_diff + (*bottom)[0]->offset(n), true);
        } else {
          caffe_gpu_backfill(offset, top_diff + offset_ng, mask + offset_ng, 
                            bottom_diff + (*bottom)[0]->offset(n) + (g * stride_ - pad_) * offset, true);
        }
      }
    }
    break;
  case TemporalPoolingParameter_PoolMethod_AVE:
    NOT_IMPLEMENTED;
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
}

INSTANTIATE_CLASS(TemporalPoolingLayer);

}  // namespace caffe
