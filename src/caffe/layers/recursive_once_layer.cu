#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
#include <algorithm>

namespace caffe {

template <typename Dtype>
void RecursiveOnceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  for (int i = 0; i < bottom.size(); ++i) {

    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = (*top)[i]->mutable_gpu_data();
    Dtype* max_idx_data = max_idx_.mutable_gpu_data();
    Dtype* weight_buf_data = weight_buffer_.mutable_gpu_data(); // reshaped weight matrix
    Dtype* out_data = tmp_buffer_.mutable_gpu_data();       // tmp output
    const Dtype* weight = this->blobs_[0]->gpu_data();

    //int weight_offset = M_ * K_;  // number of filter parameters in a group
    int across_offset = vl_ * stride_ * N_;  // number of values in an input region
    int top_offset = vl_ * N_;  // number of values in an output region / column
    int src_weight_offset = vl_ * vl_ * num_uv_;  // number of values in one set of src weight 
    
    // reshape weight matrix  
    //   -- dst weight matrix [assemble_size*vl , across*vl]
    //   -- src weight matrix [assemble_size , num_uv, vl, vl]
    for(int as = 0; as < assemble_size_; ++as) {
      for(int h = 0; h < vl_; ++h) {
        for (int wc = 0; wc < num_uv_; ++wc) {
          caffe_copy(vl_, weight+src_weight_offset * as + vl_*vl_*wc + vl_*h,
                     weight_buf_data+(as*vl_+h)*K_+vl_*relative_position_[wc]);
        }
      }
    }

    for (int n = 0; n < num_; ++n) {
      for (int g = 0; g < group_out_; ++g) {
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, K_,
            (Dtype)1., weight_buf_data , bottom_data + bottom[i]->offset(n) + across_offset * g,
            (Dtype)0., out_data);
        if (bias_term_) {
          caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, 
              N_, 1, (Dtype)1., this->blobs_[1]->gpu_data(),
              bias_multiplier_.gpu_data(),
              (Dtype)1., out_data);
        }
        // max-out to top
        Dtype * top_data_g = top_data + (*top)[i]->offset(n) + top_offset * g;
        caffe_copy(top_offset, out_data, top_data_g);
        Dtype *max_idx_data_g = max_idx_data + max_idx_.offset(n) + top_offset * g;
        caffe_set(top_offset, Dtype(0), max_idx_data_g);
        if (multi_weights_) {
          for (int nid = 1; nid < assemble_size_; ++nid) {
            caffe_vimax(top_offset, top_data_g, max_idx_data_g, out_data + top_offset * nid, static_cast<Dtype>(nid));
          }
        }
      }
    }

  }
}

template <typename Dtype>
void RecursiveOnceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {

  Dtype* tmp_diff = tmp_buffer_.mutable_gpu_diff();
  Dtype* max_idx_data = max_idx_.mutable_gpu_data();
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_set(this->blobs_[0]->count(), Dtype(0), weight_diff);
  }
  Dtype* bias_diff = NULL;
  if (bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_set(this->blobs_[1]->count(), Dtype(0), bias_diff);
  }

  Dtype* weight_buf_data = weight_buffer_.mutable_gpu_data(); // reshaped weight matrix
  Dtype* weight_buf_diff = weight_buffer_.mutable_gpu_diff(); // reshaped weight matrix diff
  caffe_set(M_ * K_, Dtype(0), weight_buf_diff);

  const int top_offset = vl_ * N_;
  int across_offset = vl_ * stride_ * N_;  // number of values in an input region
  int src_weight_offset = vl_ * vl_ * num_uv_;  // number of values in one set of src weight 

  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const Dtype* bottom_data = (*bottom)[i]->gpu_data();
    Dtype* bottom_diff = (*bottom)[i]->mutable_gpu_diff();
    caffe_set(num_*channels_*height_*width_, Dtype(0), bottom_diff);
    
    if (this->param_propagate_down_[0] || propagate_down[i] || 
                    bias_term_ && this->param_propagate_down_[1]) {
      for (int n = 0; n < num_; ++n) {
        for (int g = 0; g < group_out_; ++g) {
          // top to tmp
          int offset_ng = top[0]->offset(n) + top_offset * g;
          caffe_gpu_backfill(top_offset, top_diff + offset_ng,
                             max_idx_data + offset_ng, tmp_diff);
          // Bias gradient, if necessary.
          if (bias_term_ && this->param_propagate_down_[1]) {
            caffe_gpu_gemv<Dtype>(CblasNoTrans, vl_ * assemble_size_, N_,
                1., tmp_diff,
                bias_multiplier_.gpu_data(), 1.,
                bias_diff);
          }
          if (this->param_propagate_down_[0] || propagate_down[i]) {

            // gradient w.r.t. weight. Note that we will accumulate diffs.
            if (this->param_propagate_down_[0]) {
              caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, K_, N_,
                  (Dtype)1., tmp_diff,
                  bottom_data + (*bottom)[i]->offset(n) + g * across_offset, (Dtype)1.,
                  weight_buf_diff);
            }
            // weight_buf diff back to the weight diff 
            // reshape weight matrix  
            //   -- dst weight matrix [assemble_size*vl , across*vl]
            //   -- src weight matrix [assemble_size , num_uv, vl, vl]
            for(int as = 0; as < assemble_size_; ++as) {
              for(int h = 0; h < vl_; ++h) {
                for (int wc = 0; wc < num_uv_; ++wc) {
                  caffe_copy(vl_,
                             weight_buf_diff+(as*vl_+h)*K_+vl_*relative_position_[wc],
                             weight_diff+src_weight_offset * as + vl_*vl_*wc + vl_*h);
                }
              }
            }
            // gradient w.r.t. bottom data, if necessary.
            if (propagate_down[i]) {
              if (weight == NULL) {
                weight = this->blobs_[0]->gpu_data();
              }
              caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, K_, N_, M_,
                  (Dtype)1., weight_buf_data,
                  tmp_diff,
                  (Dtype)1., bottom_diff + (*bottom)[i]->offset(n) + g * across_offset);
            }
          }
        }
      }
    }
  }
}

INSTANTIATE_CLASS(RecursiveOnceLayer);

}  // namespace caffe
